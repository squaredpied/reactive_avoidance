#include "hip/hip_runtime.h"
#include <Eigen/QR>
#include <cmath>
#include <iostream>

#include "nvblox/gpu_hash/cuda/gpu_hash_interface.cuh"
#include "nvblox/utils/timing.h"
#include "rmpcpp_planner/policies/lidarray_CUDA.h"
#include "rmpcpp_planner/policies/misc.cuh"

#define BLOCKSIZE 16

// Lidar datapoint as output by an Ouster OS-1
typedef struct LidarPointOuster {  //     Start     End     Size
  float x;                         //        0       3         4
  float y;                         //        4       7         4
  float z;                         //        8       11        4
  uint32_t unkown_0;               //        12      15        4
  float intensity;                 //        16      19        4
  uint32_t time;                   //        20      23        4
  uint16_t reflectivity;           //        24      25        2
  uint8_t ring;                    //        26      26        1
  uint8_t unknown_1;               //        27      27        1
  uint16_t ambient;                //        28      29        2
  uint16_t unknown_2;              //        30      31        2
  uint32_t range;                  //        32      35        4
  uint32_t unknown_3;              //        36      39        4
  uint32_t unknown_4;              //        40      43        4
  uint32_t unknown_5;              //        44      47        4
  __device__ inline Eigen::Vector3f ray() {
    return {x, y, z};
    // tried doing this with Eigen::Map<> but always somehow got a warp
    // memory alignment problem. so for now leaving the copying.
  }
  __device__ inline float ray_length() { return range / 1000.0; }
} __attribute__((
    packed));  // important! we want same memory alignment as in the message


// Lidar data point as used in Gazebo Rotors
typedef struct LidarPointSim {  //     Start     End     Size
  float x;                      //        0       3         4
  float y;                      //        4       7         4
  float z;                      //        8       11        4
  float intensity;              //        12      15        4
  uint16_t ring;                //        16      17        2
  uint32_t time;                //        18      21        4

  __device__ inline Eigen::Vector3f ray() { return {x, y, z}; }
  __device__ inline float ray_length() { return ray().norm(); }
} __attribute__((
    packed));  // important! we want same memory alignment as in the message

// Should be solved nicer in the future.
#ifdef USE_OUSTER_LIDAR
  typedef LidarPointOuster LidarPoint;
#else
  typedef LidarPointSim LidarPoint;
#endif


__global__ void raycastKernel(
    const Eigen::Vector3f vel, const uint8_t* lidar_data,
    size_t lidar_data_n_points, Eigen::Matrix3f* metric_sum,
    Eigen::Vector3f* metric_x_force_sum, float maximum_ray_length,
    const RaycastingCudaPolicyParameters parameters, bool output_debug,
    rmpcpp::LidarRayDebugData* output_values,
    rmpcpp::LidarPolicyDebugData* policy_debug_data) {
  /** Shared memory to sum the components of an RMP */
  using Vector = Eigen::Vector3f;
  using Matrix = Eigen::Matrix3f;

  /** Thread ids */
  const unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
  const unsigned int dimx = gridDim.x * blockDim.x;
  const unsigned int idy = threadIdx.y + blockIdx.y * blockDim.y;
  const unsigned int dimy = gridDim.y * blockDim.y;
  const unsigned int id = idx + idy * dimx;

  LidarPoint* point = (LidarPoint*)lidar_data;

  float ray_length = point[id].ray_length();
  output_values[id].topRows<3>(0) = point[id].ray();
  Eigen::Vector3f ray = point[id].ray().normalized();

  Matrix A_obst, A;
  Vector metric_x_force;

  rmpcpp::LidarPolicyDebugData policy_data;

  if (ray_length >= maximum_ray_length || id >= lidar_data_n_points ||
      ray_length <= 0.2 || point[id].reflectivity <= 25) {
    // (here we simply also filter out some of the obvious outliers.
    // Exact numbers might depend on lidar model though.)

    /** No obstacle hit: return */
    A = Matrix::Zero();
    metric_x_force = Vector::Zero();
    policy_data = rmpcpp::LidarPolicyDebugData::Zero();

    output_values[id](3) = 0.0;
    output_values[id](4) = 0.0;
    output_values[id](5) = 0.0;
    output_values[id](6) = 0.0;
    output_values[id](7) = 0.0;
  } else {
    /** Simple RMP obstacle policy */
    Vector f_rep = alpha_rep(ray_length, parameters.eta_rep, parameters.v_rep,
                             parameters.lin_rep) *
                   -ray;
    Vector f_damp = -alpha_damp(ray_length, parameters.eta_damp,
                                parameters.v_damp, parameters.epsilon_damp) *
                    max(0.0, float(-vel.transpose() * -ray)) *
                    (-ray * -ray.transpose()) * vel;
    Vector f_obst = f_rep + f_damp;
    Vector f_norm = softnorm(f_obst, parameters.c_softmax_obstacle);

    if (parameters.metric) {
      A_obst = w(ray_length, parameters.r) * f_norm * f_norm.transpose();
    } else {
      A_obst = w(ray_length, parameters.r) * Matrix::Identity();
    }

    A = A_obst;
    Vector f = f_obst;
    metric_x_force = A * f;

    policy_data.col(0) = f_rep;
    policy_data.col(1) = f_norm;
    policy_data.col(2) = A * f_rep;
    policy_data.col(3) = A * f_norm;

    output_values[id](3) = f.norm();
    output_values[id](4) = A.norm();
    output_values[id](5) = metric_x_force.norm();
    output_values[id](6) = f_damp.norm();
    output_values[id](7) = f_rep.norm();
  }

  const int blockId = blockIdx.x + blockIdx.y * gridDim.x;

  /** Reduction within CUDA block: Start with metric reduction */
  using BlockReduceMatrix =
      typename hipcub::BlockReduce<Matrix, BLOCKSIZE, hipcub::BLOCK_REDUCE_RAKING,
                                BLOCKSIZE>;
  __shared__ typename BlockReduceMatrix::TempStorage temp_storage_matrix;
  Matrix sum_matrices0 = BlockReduceMatrix(temp_storage_matrix)
                             .Sum(A);  // Sum calculated on thread 0

  /** Metric x force reduction */
  using BlockReduceVector =
      typename hipcub::BlockReduce<Vector, BLOCKSIZE, hipcub::BLOCK_REDUCE_RAKING,
                                BLOCKSIZE>;
  __shared__ typename BlockReduceVector::TempStorage temp_storage_vector;
  Vector sum_vectors0 =
      BlockReduceVector(temp_storage_vector).Sum(metric_x_force);

  /** Reduction within CUDA block: Start with metric reduction */
  using BlockReduceDebugData =
      typename hipcub::BlockReduce<rmpcpp::LidarPolicyDebugData, BLOCKSIZE,
                                hipcub::BLOCK_REDUCE_RAKING, BLOCKSIZE>;
  __shared__ typename BlockReduceDebugData::TempStorage temp_storage_debug_data;
  rmpcpp::LidarPolicyDebugData sum_debugdata =
      BlockReduceDebugData(temp_storage_debug_data)
          .Sum(policy_data);  // Sum calculated on thread 0

  __syncthreads();
  if (threadIdx.x == 0 && threadIdx.y == 0) {
    metric_x_force_sum[blockId] = sum_vectors0;
    metric_sum[blockId] = sum_matrices0;
    policy_debug_data[blockId] = sum_debugdata;
  }
}
/********************************************************************/

template <class Space>
rmpcpp::LidarRayCudaPolicy<Space>::LidarRayCudaPolicy(
    RaycastingCudaPolicyParameters params)
    : parameters_(*dynamic_cast<RaycastingCudaPolicyParameters*>(&params)) {
  hipStreamCreate(&stream_);
  const int blockdim = parameters_.N_sqrt / BLOCKSIZE;

  hipDeviceSetLimit(hipLimitPrintfFifoSize, 1024 * 1024 * 80);

  hipHostMalloc(&metric_sum_, sizeof(Eigen::Matrix3f) * blockdim * blockdim);
  hipHostMalloc(&metric_x_force_sum_,
                 sizeof(Eigen::Vector3f) * blockdim * blockdim);
  hipHostMalloc(&policy_debug_data_,
                 sizeof(rmpcpp::LidarPolicyDebugData) * blockdim * blockdim);

  hipMalloc(&policy_debug_data_device_,
             sizeof(rmpcpp::LidarPolicyDebugData) * blockdim * blockdim);

  hipMalloc((void**)&metric_sum_device_,
             sizeof(Eigen::Matrix3f) * blockdim * blockdim);
  hipMalloc((void**)&metric_x_force_sum_device_,
             sizeof(Eigen::Vector3f) * blockdim * blockdim);
  hipMalloc(
      &lidar_data_device_,
      sizeof(LidarPoint) * (parameters_.N_sqrt * parameters_.N_sqrt + 1));

  // this is some hack (needed?) for memory alignment.
  // Basically, we malloc one struct more than needed, an then shift
  // around the initial pointer such that its memory address is aligned
  // w.r.t to the size of the structure (i.e. every thread accesses an address
  // that is  (modulo) sizeof(LidarPoint) = 0.
  // there's probably nicer ways of doing this.
  long residual = (long)lidar_data_device_ % sizeof(LidarPoint);
  lidar_data_device_ += sizeof(LidarPoint) - residual;

  if (output_results_) {
    hipMalloc(&output_cloud_,
               sizeof(LidarRayDebugData) *
                   (parameters_.N_sqrt * parameters_.N_sqrt + 1));

    // manual alignment
    output_results_ += sizeof(LidarRayDebugData) -
                       ((long)output_results_ % sizeof(LidarRayDebugData));
  }
}
template rmpcpp::LidarRayCudaPolicy<rmpcpp::Space<2>>::LidarRayCudaPolicy(
    RaycastingCudaPolicyParameters parameters);
template rmpcpp::LidarRayCudaPolicy<rmpcpp::Space<3>>::LidarRayCudaPolicy(
    RaycastingCudaPolicyParameters parameters);

template <class Space>
void rmpcpp::LidarRayCudaPolicy<Space>::cudaStartEval(const PState& state) {
  throw std::logic_error("Not implemented");
};

/**
 * Start evaluation. Only implemented for 3d worlds
 * @param state
 */
template <>
void rmpcpp::LidarRayCudaPolicy<rmpcpp::Space<3>>::cudaStartEval(
    const PState& state) {
  /** State vectors */
  Eigen::Vector3f pos = state.pos_.cast<float>();
  Eigen::Vector3f vel = state.vel_.cast<float>();

  const int blockdim = parameters_.N_sqrt / BLOCKSIZE;

  constexpr dim3 kThreadsPerThreadBlock(BLOCKSIZE, BLOCKSIZE, 1);
  const dim3 num_blocks(blockdim, blockdim, 1);

  raycastKernel<<<num_blocks, kThreadsPerThreadBlock, 0, stream_>>>(
      vel, lidar_data_device_, lidar_data_n_points_, metric_sum_device_,
      metric_x_force_sum_device_, parameters_.r * 10, parameters_, true,
      output_cloud_, policy_debug_data_device_);
}

template <>
void rmpcpp::LidarRayCudaPolicy<rmpcpp::Space<3>>::updateLidarData(
    const LidarData& lidar_data) {
  if (lidar_data.n_points > parameters_.N_sqrt * parameters_.N_sqrt) {
    std::cout << "WARNING POINT CLOUD TRUNCATION - got point cloud with "
              << lidar_data.n_points << " points" << std::endl;
  }

  if (sizeof(LidarPoint) != lidar_data.stride) {
    std::cout << "Point Stride mismatch, ignoring data." << std::endl;
    lidar_data_n_points_ = 0;
    return;
  }

  size_t data_to_copy =
      std::min(lidar_data.size, (unsigned long)parameters_.N_sqrt *
                                    parameters_.N_sqrt * sizeof(LidarPoint));
  lidar_data_n_points_ =
      std::min(lidar_data.n_points,
               (unsigned long)parameters_.N_sqrt * parameters_.N_sqrt);

  // copy in new data.
  hipMemcpy(lidar_data_device_, (void*)lidar_data.data, data_to_copy,
             hipMemcpyHostToDevice);
}

/**
 * Blocking call to evaluate at state.
 * @param state
 * @return
 */
template <>
rmpcpp::LidarRayCudaPolicy<rmpcpp::Space<3>>::PValue
rmpcpp::LidarRayCudaPolicy<rmpcpp::Space<3>>::evaluateAt(const PState& state) {
  const int blockdim = parameters_.N_sqrt / BLOCKSIZE;

  if (!async_eval_started_) {
    cudaStartEval(state);
  }
  /** If an asynchronous eval was started, no check is done whether the state is
   * the same. (As for now this should never happen)*/
  hipStreamSynchronize(stream_);
  async_eval_started_ = false;

  hipMemcpy(metric_sum_, metric_sum_device_,
             sizeof(Eigen::Matrix3f) * blockdim * blockdim,
             hipMemcpyDeviceToHost);
  hipMemcpy(metric_x_force_sum_, metric_x_force_sum_device_,
             sizeof(Eigen::Vector3f) * blockdim * blockdim,
             hipMemcpyDeviceToHost);

  hipMemcpy(policy_debug_data_, policy_debug_data_device_,
             sizeof(rmpcpp::LidarPolicyDebugData) * blockdim * blockdim,
             hipMemcpyDeviceToHost);

  Eigen::Matrix3f sum = Eigen::Matrix3f::Zero();
  Eigen::Vector3f sumv = Eigen::Vector3f::Zero();
  rmpcpp::LidarPolicyDebugData debug_data;

  for (int i = 0; i < blockdim * blockdim; i++) {
    sum += metric_sum_[i];
    sumv += metric_x_force_sum_[i];

    debug_data += policy_debug_data_[i];
  }
  if (sum.isZero(0.001)) {  // Check if not all values are 0, leading to
                            // unstable inverse
    return {Vector::Zero(), Matrix::Zero()};
  }

  Eigen::Matrix3d sumd = sum.cast<double>();
  Eigen::Matrix3d sumd_inverse =
      sumd.completeOrthogonalDecomposition().pseudoInverse();

  Eigen::Vector3d f = sumd_inverse * sumv.cast<double>();
  last_evaluated_state_.pos_ = state.pos_;
  last_evaluated_state_.vel_ = state.vel_;

  // recover debug policies
  debug_data.col(0) /=
      (parameters_.N_sqrt * parameters_.N_sqrt);  // no metric, just average
  debug_data.col(1) /=
      (parameters_.N_sqrt * parameters_.N_sqrt);  // no metric, just average
  debug_data.col(2) = sumd_inverse.cast<float>() * debug_data.col(2);
  debug_data.col(3) = sumd_inverse.cast<float>() *
                      debug_data.col(3);  // multiply by inverse metric

  return {f, sumd};
}

/**
 * Not implemented for 2d
 * @param state
 * @return
 */
template <>
rmpcpp::LidarRayCudaPolicy<rmpcpp::Space<2>>::PValue
rmpcpp::LidarRayCudaPolicy<rmpcpp::Space<2>>::evaluateAt(const PState& state) {
  std::cout << "LIDARRAYCUDAPOLICY 2 called" << std::endl;
  throw std::logic_error("Not implemented");
}

/**
 * Starts asynchronous evaluation (so returns before it is done)
 * @tparam Space
 * @param state
 */
template <class Space>
void rmpcpp::LidarRayCudaPolicy<Space>::startEvaluateAsync(
    const PState& state) {
  cudaStartEval(state);
  async_eval_started_ = true;
}
template void rmpcpp::LidarRayCudaPolicy<rmpcpp::Space<2>>::startEvaluateAsync(
    const PState& state);
template void rmpcpp::LidarRayCudaPolicy<rmpcpp::Space<3>>::startEvaluateAsync(
    const PState& state);

/**
 * Abort asynchronous evaluation
 * @tparam Space
 */
template <class Space>
void rmpcpp::LidarRayCudaPolicy<Space>::abortEvaluateAsync() {
  hipStreamSynchronize(stream_);
  hipFree(metric_sum_device_);
  hipFree(metric_x_force_sum_device_);
  async_eval_started_ = false;
}
template void
rmpcpp::LidarRayCudaPolicy<rmpcpp::Space<2>>::abortEvaluateAsync();
template void
rmpcpp::LidarRayCudaPolicy<rmpcpp::Space<3>>::abortEvaluateAsync();
